#define n_train 60000
#define n_test  10000
#define tmax 256
#define Imax 255
//int n_layers = 2;
const int img_size   = 28*28;
const int n_inp      = 28*28;
const int n_hid      = 1000;
const int n_out      = 10;
const int n_epochs   = 1;
const int batch_size = 100;
const int n_batches  = n_train/batch_size;
const int th_val     = 100;
const int a1 = 0, b1 = 5, a2 = 0, b2 = 50;

#include <stdio.h>
#include <stdlib.h>
#include "mnist_load.cpp"
#include "encoding.cpp"
#include "hip/hip_runtime.h"
#include "initialization.cpp"
#include "forward_pass_cuda.cu"

int main(int argc, char** argv){

	float *w1, *w2, *dev_w1, *dev_w2;

	w1 = (float*) malloc(sizeof(float)*img_size*n_hid);
    w2 = (float*) malloc(sizeof(float)*n_hid*n_out);

	hipMalloc((void **) &dev_w1, sizeof(float)*img_size*n_hid);
    hipMalloc((void **) &dev_w2, sizeof(float)*n_hid*n_out);
	
	weight_init_1D(w1, w2, img_size*n_hid, n_hid*n_out);

	hipMemcpy(dev_w1, w1, sizeof(float)*n_inp*n_hid, hipMemcpyHostToDevice);
	hipMemcpy(dev_w2, w2, sizeof(float)*n_hid*n_out, hipMemcpyHostToDevice);

	vector<vector<double>> x_train,  x_test;
	vector<int> y_train, y_test;

	ReadMNIST(60000,784,x_train,"../train-images-idx3-ubyte");
	ReadMNIST(10000,784,x_test,"../t10k-images-idx3-ubyte");
	ReadMNIST_label(60000,  y_train,"../train-labels-idx1-ubyte");
	ReadMNIST_label(10000,  y_test,"../t10k-labels-idx1-ubyte");

	cout <<"Training the model....\n";
	//Training:
	for (int e = 0; e < n_epochs; ++e)
	{
		cout <<"epoch: "<<e+1<<" \n";
		rand_shuffle(x_train, y_train);

		for (int b = 0; b < n_batches; ++b)
		{
			cout<<"batch: "<<b+1<<"\n";

			int *x_batch, *dev_x_batch, *dev_x1_out, *dev_x2_out, *firing_t0, *dev_firing_t0, *dev_firing_t1, *dev_firing_t2;
			x_batch = (int *) calloc(batch_size*n_inp*tmax, sizeof(int));
			firing_t0 = (int *) malloc(sizeof(int)*batch_size*n_inp);
			vector<int> y_batch; // (batch_size, n_out)
			
			hipMalloc((void **) &dev_x_batch, sizeof(int)*batch_size*n_inp*tmax);
			hipMalloc((void **) &dev_x1_out, sizeof(int)*batch_size*n_hid*tmax);
			hipMalloc((void **) &dev_x2_out, sizeof(int)*batch_size*n_out*tmax);
			hipMalloc((void **) &dev_firing_t0, sizeof(int)*batch_size*n_inp);
			hipMalloc((void **) &dev_firing_t1, sizeof(int)*batch_size*n_hid);
			hipMalloc((void **) &dev_firing_t2, sizeof(int)*batch_size*n_out);
			hipMemset(dev_x1_out, 0, sizeof(int)*batch_size*n_hid*tmax);
			hipMemset(dev_x1_out, 0, sizeof(int)*batch_size*n_out*tmax);

			batch_spike_encoding_1D(x_train, y_train, x_batch, y_batch, firing_t0, b, batch_size, n_inp);

			hipMemcpy(dev_x_batch, x_batch, sizeof(int)*batch_size*n_inp*tmax, hipMemcpyHostToDevice);
			hipMemcpy(dev_firing_t0, firing_t0, sizeof(int)*batch_size*n_inp, hipMemcpyHostToDevice);

			batch_dense(dev_x_batch, dev_x1_out, dev_firing_t1, dev_w1, n_inp, n_hid, batch_size, th_val, tmax);
			batch_dense(dev_x1_out, dev_x2_out, dev_firing_t2, dev_w2, n_hid, n_out, batch_size, th_val, tmax);
		

	// 		//loss_calc(x2_out, y_batch, grad2);

	// 		//backprop_dense(grad2, w2, grad1, lr2);
	// 		//backprop_dense(grad1, w1, grad0, lr1);

	// 		/*
	// 		//backprop(grad2, w2, grad1, lr2, dw2);
	// 		//update_weights(w2, dw1, lr2);
	// 		*/

	        free(x_batch);
			free(firing_t0);
			hipFree(dev_x_batch);
	        hipFree(dev_x1_out);
	        hipFree(dev_x2_out);
			hipFree(dev_firing_t0);
			hipFree(dev_firing_t1);
			hipFree(dev_firing_t2);
		}
	}

    free(w1);
	free(w2);
	hipFree(dev_w1);
	hipFree(dev_w2);
	return 0;
	
}


