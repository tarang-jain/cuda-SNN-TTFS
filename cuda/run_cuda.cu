#include "hip/hip_runtime.h"
#define n_train 60000
#define n_test  10000
#define tmax 256
#define Imax 255
//int n_layers = 2;
const int img_size   = 28*28;
const int n_inp      = 28*28;
const int n_hid      = 400;
const int n_out      = 10;
const int n_epochs   = 30;
const int batch_size = 100;
const int n_batches  = n_train / batch_size;
const int th_val     = 100;
const int gamma_     = 3;
const double lr1        = 0.2;
const double lr2        = 0.2;
const int a1 = 0, b1 = 5, a2 = 0, b2 = 50;

#include <stdio.h>
#include <stdlib.h>
#include "mnist_load.cpp"
#include "encoding.cpp"
#include "hip/hip_runtime.h"
#include "initialization.cpp"
#include "forward_pass_cuda.cu"
#include "backward_pass_cuda.cu"
int main(int argc, char** argv){

	int *x_batch, * dev_x_batch, *dev_x1_out, *dev_x2_out, *firing_t0, *dev_firing_t0, *dev_firing_t1, *dev_firing_t2, *y_batch, *dev_y_batch;
	double *w1, *w2, *dev_w1, *dev_w2, *dev_dw1, *dev_dw2, *dev_delta0, *dev_delta1, *dev_delta2, *dev_correct, *dev_lr1, *dev_lr2;

	w1 = (double*) malloc(sizeof(double)*img_size*n_hid);
    w2 = (double*) malloc(sizeof(double)*n_hid*n_out);
	x_batch = (int *) malloc(sizeof(int)*batch_size*n_inp*tmax);
	firing_t0 = (int *) malloc(sizeof(int)*batch_size*n_inp);
	y_batch = (int *) malloc(sizeof(int)*batch_size);
	hipMalloc((void **) &dev_w1, sizeof(double)*img_size*n_hid);
    hipMalloc((void **) &dev_w2, sizeof(double)*n_hid*n_out);
	hipMalloc((void **) &dev_dw1, sizeof(double)*img_size*n_hid);
    hipMalloc((void **) &dev_dw2, sizeof(double)*n_hid*n_out);
	hipMalloc((void **) &dev_x_batch, sizeof(int)*batch_size*n_inp*tmax);
	hipMalloc((void **) &dev_x1_out, sizeof(int)*batch_size*n_hid*tmax);
	hipMalloc((void **) &dev_x2_out, sizeof(int)*batch_size*n_out*tmax);
	hipMalloc((void **) &dev_firing_t0, sizeof(int)*batch_size*n_inp);
	hipMalloc((void **) &dev_firing_t1, sizeof(int)*batch_size*n_hid);
	hipMalloc((void **) &dev_firing_t2, sizeof(int)*batch_size*n_out);
	hipMalloc((void **) &dev_delta0, sizeof(double)*batch_size*n_inp);
	hipMalloc((void **) &dev_delta1, sizeof(double)*batch_size*n_hid);
	hipMalloc((void **) &dev_delta2, sizeof(double)*batch_size*n_out);
	hipMalloc((void **) &dev_correct, sizeof(int));
	hipMalloc((void **) &dev_lr1, sizeof(double));
	hipMalloc((void **) &dev_lr2, sizeof(double));
	hipMalloc((void **) &dev_y_batch, sizeof(int)*batch_size);

	weight_init_1D(w1, w2, img_size*n_hid, n_hid*n_out);

	hipMemcpy(dev_w1, w1, sizeof(double)*n_inp*n_hid, hipMemcpyHostToDevice);
	hipMemcpy(dev_w2, w2, sizeof(double)*n_hid*n_out, hipMemcpyHostToDevice);
	hipMemcpy(dev_lr1, &lr1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_lr2, &lr2, sizeof(double), hipMemcpyHostToDevice);

	vector<vector<double>> x_train,  x_test;
	vector<int> y_train, y_test;

	ReadMNIST(60000,784,x_train,"../train-images-idx3-ubyte");
	ReadMNIST(10000,784,x_test,"../t10k-images-idx3-ubyte");
	ReadMNIST_label(60000,  y_train,"../train-labels-idx1-ubyte");
	ReadMNIST_label(10000,  y_test,"../t10k-labels-idx1-ubyte");



	cout <<"Training the model....\n";
	//Training:
	for (int e = 0; e < n_epochs; ++e)
	{
		cout <<"epoch: "<<e+1<<" \n";
		// rand_shuffle(x_train, y_train);

		int training_acc = 0;
		int *dev_training_acc;

		hipMalloc((void **) &dev_training_acc, sizeof(int));
		hipMemcpy(dev_training_acc, &training_acc, sizeof(int), hipMemcpyHostToDevice);

		for (int b = 0; b < n_batches; ++b)
		{
			cout<<"batch: "<<b+1<<"\n";




			memset(x_batch, 0, batch_size*n_inp*tmax);

			hipMemset(dev_dw1, 0, sizeof(double)*img_size*n_hid);
			hipMemset(dev_dw2, 0, sizeof(double)*n_hid*n_out);			
			hipMemset(dev_x1_out, 0, sizeof(int)*batch_size*n_hid*tmax);
			hipMemset(dev_x1_out, 0, sizeof(int)*batch_size*n_out*tmax);

			batch_spike_encoding_1D(x_train, y_train, x_batch, y_batch, firing_t0, b, batch_size, n_inp);

			hipMemcpy(dev_x_batch, x_batch, sizeof(int)*batch_size*n_inp*tmax, hipMemcpyHostToDevice);
			hipMemcpy(dev_y_batch, y_batch, sizeof(int)*batch_size, hipMemcpyHostToDevice);
			hipMemcpy(dev_firing_t0, firing_t0, sizeof(int)*batch_size*n_inp, hipMemcpyHostToDevice);

			batch_dense(dev_x_batch, dev_x1_out, dev_firing_t1, dev_w1, n_inp, n_hid, batch_size, th_val, tmax);
			batch_dense(dev_x1_out, dev_x2_out, dev_firing_t2, dev_w2, n_hid, n_out, batch_size, th_val, tmax);

			dim3 threadsPerBlock_2D(n_out, batch_size);
			dim3 blocksPerGrid_2D(1, 1);


			if (batch_size > floor(double(1024)/double(n_out))){
				threadsPerBlock_2D.y = floor(double(1024)/double(n_out));
				blocksPerGrid_2D.y = ceil(double(batch_size)/double(threadsPerBlock_2D.y));
			}


			loss_calc<<<blocksPerGrid_2D, threadsPerBlock_2D>>>(dev_firing_t2, dev_y_batch, dev_delta2, batch_size, n_out, tmax, dev_training_acc);

			hipMemcpy(&training_acc, dev_training_acc, sizeof(int), hipMemcpyDeviceToHost);

			cout<<"\t\ttrain acc = "<<training_acc<<"/"<<((b+1)*batch_size)<<" = "<<(training_acc/(1.0*(b+1)*batch_size))<<"\n";

			backprop_dense(dev_firing_t2, dev_firing_t1, dev_delta2, dev_delta1, dev_w2, dev_dw2, lr2, true, batch_size, n_hid, n_out);

			if (n_hid > floor(double(1024)/double(n_out))){
				threadsPerBlock_2D.y = floor(double(1024)/double(n_out));
				blocksPerGrid_2D.y = ceil(double(n_hid)/double(threadsPerBlock_2D.y));
			}

			update_weights<<<blocksPerGrid_2D, threadsPerBlock_2D>>>(dev_w2, dev_dw2, lr2, n_hid, n_out);

			backprop_dense(dev_firing_t1, dev_firing_t0, dev_delta1, dev_delta0, dev_w1, dev_dw1, lr1, false, batch_size, n_inp, n_hid);

			threadsPerBlock_2D.x = n_hid;
			threadsPerBlock_2D.y = n_inp;

			if (n_inp > floor(double(1024)/double(n_hid))){
				threadsPerBlock_2D.y = floor(double(1024)/double(n_hid));
				blocksPerGrid_2D.y = ceil(double(n_inp)/double(threadsPerBlock_2D.y));
			}

			update_weights<<<blocksPerGrid_2D, threadsPerBlock_2D>>>(dev_w1, dev_dw1, lr1, n_inp, n_hid);

		}
	}

	free(x_batch);
	free(firing_t0);
	free(w1);
	free(w2);
	hipFree(dev_x_batch);
	hipFree(dev_x1_out);
	hipFree(dev_x2_out);
	hipFree(dev_firing_t0);
	hipFree(dev_firing_t1);
	hipFree(dev_firing_t2);
	hipFree(dev_w1);
	hipFree(dev_w2);
	hipFree(dev_dw1);
	hipFree(dev_dw2);
	hipFree(dev_dw2);
	hipFree(dev_delta0);
	hipFree(dev_delta1);
	hipFree(dev_delta2);
	hipFree(dev_correct);
	hipFree(dev_lr1);
	hipFree(dev_lr2);

	return 0;
	
}


