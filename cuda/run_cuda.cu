#define n_train 60000
#define n_test  10000
#define tmax 256
#define Imax 255

//int n_layers = 2;
const int img_size   = 28*28;
const int n_inp      = 28*28;
const int n_hid      = 10;
const int n_out      = 10;
const int n_epochs   = 1;
const int batch_size = 100;
const int n_batches  = n_train/batch_size;
const int th_val     = 100;
const int a1 = 0, b1 = 5, a2 = 0, b2 = 50;

#include <hip/hip_runtime.h>
#include "mnist_load.cpp"
#include "encoding_cuda.cpp"
#include "initialization_cuda.cpp"
#include "forward_pass_cuda.cu"

int main(int argc, char** argv){

	float *w1, *w2, *dev_w1, *dev_w2;

	// w1 = (float*) malloc(sizeof(float)*img_size*n_hid);
    // w2 = (float*) malloc(sizeof(float)*n_hid*n_out);

	hipMalloc((void **) &dev_w1, sizeof(float)*img_size*n_hid);
    hipMalloc((void **) &dev_w2, sizeof(float)*n_hid*n_out);
	
	weight_init_cuda(dev_w1, img_size*n_hid);
	weight_init_cuda(dev_w2, n_hid*n_out);

	vector<vector<double>> x_train,  x_test;
	vector<int> y_train, y_test;
	//cout<<std::filesystem::exists("train-images-idx3-ubyte");
	ReadMNIST(60000,784,x_train,"../train-images-idx3-ubyte");
	ReadMNIST(10000,784,x_test,"../t10k-images-idx3-ubyte");
	ReadMNIST_label(60000,  y_train,"../train-labels-idx1-ubyte");
	ReadMNIST_label(10000,  y_test,"../t10k-labels-idx1-ubyte");

	//preproc(x_train, x_test);

	cout <<"Training the model....\n";
	//Training:
	for (int e = 0; e < n_epochs; ++e)
	{
		cout <<"epoch: "<<e+1<<" \n";
		// rand_shuffle(x_train, y_train);

		for (int b = 0; b < n_batches; ++b)
		{
			cout<<"batch: "<<b+1<<"\n";
			// vector<vector<vector<int>>> x_batch, x1_out, x2_out;
			int *x_batch, *dev_x_batch, *dev_x1_out, *dev_x2_out, *firing_t1, *firing_t2;
			x_batch = (int *) calloc(batch_size*n_inp*tmax, sizeof(int));
			vector<int> y_batch; // (batch_size, n_out)
			
			batch_spike_encoding(x_train, y_train, x_batch, y_batch, b, batch_size, n_inp);

			hipMalloc((void **) &dev_x_batch, sizeof(int)*batch_size*n_inp*tmax);
			hipMalloc((void **) &dev_x1_out, sizeof(int)*batch_size*n_hid*tmax);
			hipMalloc((void **) &dev_x2_out, sizeof(int)*batch_size*n_out*tmax);
			hipMalloc((void **) &firing_t1, sizeof(int)*batch_size*n_hid);
			hipMalloc((void **) &firing_t2, sizeof(int)*batch_size*n_out);
			hipMemset(dev_x1_out, 0, sizeof(int)*batch_size*n_hid*tmax);
			hipMemset(dev_x1_out, 0, sizeof(int)*batch_size*n_out*tmax);

			hipMemcpy(dev_x_batch, x_batch, sizeof(int)*batch_size*n_inp*tmax, hipMemcpyHostToDevice);
			batch_dense(dev_x_batch, dev_x1_out, firing_t1, dev_w1, n_inp, n_hid, batch_size, th_val, tmax);
			batch_dense(dev_x1_out, dev_x2_out, firing_t2, dev_w2, n_hid, n_out, batch_size, th_val, tmax);
		

	// 		//loss_calc(x2_out, y_batch, grad2);

	// 		//backprop_dense(grad2, w2, grad1, lr2);
	// 		//backprop_dense(grad1, w1, grad0, lr1);

	// 		/*
	// 		//backprop(grad2, w2, grad1, lr2, dw2);
	// 		//update_weights(w2, dw1, lr2);
	// 		*/
	        free(x_batch);
			hipFree(dev_x_batch);
	        hipFree(dev_x1_out);
	        hipFree(dev_x2_out);
			hipFree(firing_t1);
			hipFree(firing_t2);
		}
	}

    // free(w1);
	// free(w2);
	hipFree(dev_w1);
	hipFree(dev_w2);
	return 0;

	
}


