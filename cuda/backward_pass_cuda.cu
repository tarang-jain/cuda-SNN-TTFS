#include "hip/hip_runtime.h"
// __device__ int min(int a, int b){
    // return a <= b ? a : b;
// }

__global__ void loss_calc(int * firing_t, int * y_batch,double * delta, int batchSize, int N, int t_max, int * correct){

    int index = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int min_time[100];
    __shared__ int target[1000];
    __shared__ int winner[100];

    if (index < batchSize && row < N)
    {
        target[index*N + row] = firing_t[index*N + row];
        __syncthreads();
        if (row == 0){
            winner[index] = 0;
            min_time[index] = t_max;

            for (int i = 0; i < N; i++)
            {
                int time = firing_t[index*N + i];
                // printf("%d\n", time);
                winner[index] = time < min_time[index] ? i : winner[index];
                min_time[index] = time < min_time[index] ? time : min_time[index];
                // printf("%d\n", min_time[index]);
            }

            // printf("%d\n", y_batch[index]);
            // printf("Okay0");


            if(winner[index]==y_batch[index]) atomicAdd(correct, 1);
            // printf("%d\n", *correct);
            // printf("Okay");
        }
        __syncthreads();
        // printf("Yes\n");


        if(min_time[index] == t_max)
        {
            // printf("No\n");
            target[index*N + row] = t_max;

            if (row == y_batch[index])
            {
                target[index*N + row] = t_max - gamma_;
            }
        }
        else
        {
            // target[index*N + row] = firing_t[index*N + row];
            target[index*N + row] = (firing_t[index*N + row] - min_time[index]) < gamma_ ? (min(min_time[index] + gamma_, t_max)) : firing_t[index*N + row];
            // printf("%d\n", y_batch[index]);
            if (row == y_batch[index])
            {
                target[index*N + row] = min_time[index];
            }
        }
        // __syncthreads();

        delta[index * N + row] = double(target[index * N + row] - firing_t[index * N + row]) / double(t_max); //delta = target - firing_t;
        // printf("%d\t%d\t%f\n", index, row, delta[index*N + row]);
    }
}

__global__ void compute_norm(double * delta, double * norm, int batchSize, int N){

    int index = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < batchSize && row < N && threadIdx.x == 0){
        double sum = 0;
        for(int i = 0; i < blockDim.x; i++){ 
            sum += delta[index*N + i]*delta[index*N + i];

        atomicAdd(&norm[index], sum);
    }
}
}

__global__ void grad_norm(double * delta, double * norm, int batchSize, int N){

    int index = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    norm[index] = sqrt(norm[index]);

    if (index < batchSize && row < N){
        delta[index*N + row] /= norm[index];
    }
}

__global__ void batch_bcast_dot(double * delta, bool * fired_before, double * dw_batch, int batchSize, int N_in, int N_out){

    int index = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < batchSize && row < N_in && col < N_out)
    {
        // printf("%lf\n", delta[index*N_out + col]);
        dw_batch[index*N_in*N_out + row*N_out + col] = delta[index*N_out + col]*fired_before[index*N_in*N_out + row*N_out + col];
    }
}

__global__ void reduce_sum3d(double * dw_batch, double * dw, int batchSize, int N_in, int N_out){

    int index = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (index == 0 && row < N_in && col < N_out)
    {
        for (int b = 0; b < batchSize; ++b)
        {
            dw[row*N_out + col] += dw_batch[b*N_in*N_out + row*N_out + col];
        }
        // printf("%d\t%d\t%lf\n", row, col, dw[row*N_out + col]);  
    }  
}

__global__ void batch_multiply(double* dw_batch, double* w, double* delta, int batchSize, int N_in, int N_out){
    
    int index = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < batchSize && row < N_in && col ==0)
    {
        for (int j = 0; j < N_out; ++j)
        {
            delta[index*N_in + row] += dw_batch[index*N_in*N_out + row*N_out + j]*w[row*N_out + j];
        }  
    }  

}

__global__ void create_fired_before(int * firing_t_in, int * firing_t_out, int batchSize, bool * fired_before, int N_in, int N_out){

    int index = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    fired_before[index*N_in*N_out + row*N_out + col] = firing_t_in[index*N_in + row] < firing_t_out[index*N_out + col];    
}

//backprop_dense(firing_t1, firing_t0, delta2, delta1, w2,  lr2);
void backprop_dense(int * firing_t_out, int * firing_t_in, double * delta2,double * delta1, double * w2, double * dw2, double lr, bool calc_delta, int batchSize, int N_in, int N_out){

    double * norm, * dw2_batch;
    bool * fired_before;
    hipMalloc((void **) &norm, sizeof(double)*batchSize);
    hipMalloc((void **) &dw2_batch, sizeof(double)*batchSize*N_in*N_out);
    hipMalloc((void **) &fired_before, sizeof(bool)*batchSize*N_in*N_out);

    dim3 threadsPerBlock_2D(10, 100);
    dim3 blocksPerGrid_2D(1, 1);

    blocksPerGrid_2D.x = ceil(double(N_out)/double(threadsPerBlock_2D.x));
    blocksPerGrid_2D.y = ceil(double(batchSize)/double(threadsPerBlock_2D.y));

    compute_norm<<<blocksPerGrid_2D, threadsPerBlock_2D>>>(delta2, norm, batchSize, N_out);
    grad_norm<<<blocksPerGrid_2D, threadsPerBlock_2D>>>(delta2, norm, batchSize, N_out);

    dim3 threadsPerBlock_3D(8, 16, 8);
    dim3 blocksPerGrid_3D(1, 1, 1);

    blocksPerGrid_3D.x = ceil(double(N_out)/double(threadsPerBlock_3D.x));
    blocksPerGrid_3D.y = ceil(double(N_in)/double(threadsPerBlock_3D.y));
    blocksPerGrid_3D.z = ceil(double(batchSize)/double(threadsPerBlock_3D.z));

    create_fired_before<<<blocksPerGrid_3D, threadsPerBlock_3D>>>(firing_t_in, firing_t_out, batchSize, fired_before, N_in, N_out);
    batch_bcast_dot<<<blocksPerGrid_3D, threadsPerBlock_3D>>>(delta2, fired_before, dw2_batch, batchSize, N_in, N_out);
    reduce_sum3d<<<blocksPerGrid_3D, threadsPerBlock_3D>>>(dw2_batch, dw2, batchSize, N_in, N_out);

    if(calc_delta) batch_multiply<<<blocksPerGrid_3D, threadsPerBlock_3D>>>(dw2_batch, w2, delta1, batchSize, N_in, N_out);

}

__global__ void update_weights(double * w, double * dw, double lr, int N_in, int N_out){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    w[row*N_out + col] += lr*dw[row*N_out + col];
}