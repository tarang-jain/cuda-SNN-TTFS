#include "hip/hip_runtime.h"
int loss_calc(int * firing_t, int * y_batch,float * delta, int batchSize, int N, int t_max, int * correct){

    int index = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int min_time[batchSize];
    vector<vector<int>> target = firing_t;
    *correct = 0;

    if (index < batchSize && row < N)
    {
        if (row == 0){
            int winner = 0;
            min_time[index] = t_max;
            for (int i = 0; i < N; i++)
            {
                int time = firing_t[index*N + i];
                winner = time < min_time[index] ? i : winner;
                min_time[index] = time < min_time[index] ? time : min_time[index];
            }
            if(winner==y_batch[index]) correct++;
        }
        __syncthreads();

        if(min_time[index]==t_max)
        {
            target[index*N + row] = min_time[index];
            if (row == y_batch[index])
            {
                target[index*N + row] = min_time[index] - gamma_;
            }
        }
        else
        {
            target[index*N + row] = firing_t[index*N + row];
            target[index*N + row] = (firing_t[b][i] - min_time) < gamma_ ? (min(min_time + gamma_, tmax)) : firing_t[index*N + row];

            if (row == y_batch[index])
            {
                target[index*N + row] = min_time[index];
            }
        }

        delta[index * N + row] = target[index * N + row] - firing_t[index * N + row]; //delta = target - firing_t;
    }
}

void compute_norm(float * delta, float * norm, int batchSize, int N){

    int index = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < batchSize && row < N && threadIdx.x == 0){
        float sum = 0;
        for(int i = 0; i < blockDim.x; i++) sum += pow(delta[index*N + i], 2);
        atomicAdd(norm[index], sum);
    }
}

void grad_norm(float * delta, float * norm, int batchSize, int N){

    int index = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < batchSize && row < N){
        delta[index*N + row] /= norm[index];
    }
}

void batch_bcast_dot(float * delta, bool * fired_before, float * dw_batch){

    int index = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int b_size = fired_before.size();
    int Nin    = fired_before[0].size();
    int Nout   = fired_before[0][0].size();

    for (int b = 0; b < b_size; ++b)
    {
        for (int i = 0; i < Nin; ++i)
        {
            for (int j = 0; j < Nout; ++j)
            {
                dw_batch[b][i][j] = delta[b][j]*fired_before[b][i][j];
            }
        }
    }
}

void reduce_sum3d(const vector<vector<vector<double>>>& dw_batch, vector<vector<double>>& dw){
    //reduce_sum along axis 0
    int b_size = dw_batch.size();
    int Nin    = dw_batch[0].size();
    int Nout   = dw_batch[0][0].size();

    dw = vector<vector<double>>(Nin, vector<double>(Nout, 0));

    for (int b = 0; b < b_size; ++b)
    {
        for (int i = 0; i < Nin; ++i)
        {
            for (int j = 0; j < Nout; ++j)
            {
                dw[i][j] += dw_batch[b][i][j];
            }
        }
    }
}

void batch_multiply(const vector<vector<vector<double>>>& dw_batch, const vector<vector<double>>& w, vector<vector<double>>& delta){
    int b_size = dw_batch.size();
    int Nin = w.size();
    int Nout = w[0].size();

    //vector<vector<vector<double>>> delta_batch(b_size, vector<vector<bool>>(Nin, vector<bool>(Nout, false)));
    delta = vector<vector<double>>(b_size, vector<double>(Nin, 0));

    for (int b = 0; b < b_size; ++b)
    {
        for (int i = 0; i < Nin; ++i)
        {
            for (int j = 0; j < Nout; ++j)
            {
                //delta_batch[b][i][j] = dw_batch[b][i][j]*dw[i][j];
                delta[b][i] += dw_batch[b][i][j]*w[i][j];
            }
        }
    }
    

}

//backprop_dense(firing_t1, firing_t0, delta2, delta1, w2,  lr2);
void backprop_dense(const vector<vector<int>>& firing_t1, const vector<vector<int>>& firing_t0, vector<vector<double>>& delta2,
                    vector<vector<double>>& delta1, vector<vector<double>>& w2, vector<vector<double>>& dw2, int lr, bool calc_delta){

    int b_size = firing_t0.size();
    int Nin = w2.size();
    int Nout = w2[0].size();
    grad_norm(delta2);

    vector<vector<vector<bool>>> fired_before(b_size, vector<vector<bool>>(Nin, vector<bool>(Nout, false)));
    
    for (int b = 0; b < b_size; ++b)
    {
        for (int i = 0; i < Nin; ++i)
        {
            for (int j = 0; j < Nout; ++j)
            {
                fired_before[b][i][j] = firing_t0[b][i] < firing_t1[b][j];
            }
        }
    }
    vector<vector<vector<double>>> dw2_batch(b_size, vector<vector<double>>(Nin, vector<double>(Nout, 0)));
    //vector<vector<double>> dw2; //(Nin, vector<double>(Nout, 0));
    batch_bcast_dot(delta2, fired_before, dw2_batch);
    reduce_sum3d(dw2_batch, dw2);


    if(calc_delta) batch_multiply(dw2_batch, w2, delta1); //Check if this has to be dw2_batch then reduction

}

void update_weights(vector<vector<double>>& w, const vector<vector<double>>& dw, double lr){
    int Nin = w.size();
    int Nout = w[0].size();

    for (int i = 0; i < Nin; ++i)
    {
        for (int j = 0; j < Nout; ++j)
        {
            w[i][j] += lr*dw[i][j];
        }
    }
}
