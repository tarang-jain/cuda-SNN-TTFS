
#include <hip/hip_runtime.h>
__global__ void batch_tensordot(int * x_in, float * w, float * v, int batchSize, int N_in, int N_out, int t_max)
{
    
    int index = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    if (index < batchSize && row < N_out && col < t_max)
    {

        for (int j = 0; j < N_in; j++)
        {
	        sum += x_in[index*N_in*t_max + j*t_max + col] * w[j*N_out + row];
        }
        v[index*N_out*t_max + row*t_max + col] = sum;
    }
}

__global__ void batch_cumsum(float * v, int N_out, int t_max, int batchSize)
{
    int index = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < batchSize && row < N_out && col == 0){

    for (int j = 1; j < t_max; j++){
        v[index*N_out*t_max + row*t_max + j] += v[index*N_out*t_max + row*t_max + j-1];
    }
    }
    
}

__global__ void batch_thresholding(int batchSize, int * x, int * firing_t, float * v, int N_out, int t_max, float th_val)
{
	int index = blockIdx.z * blockDim.z + threadIdx.z;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < batchSize && row < N_out && col == 0){
		
	for (int j = 0; j < t_max; j++){
           if (v[index*N_out*t_max + row*t_max + j] >= th_val){
              firing_t[index*N_out + row] = j;
              x[index*N_out*t_max + row*t_max + j] = 1;
              break;
              
           }
		   
	           
	}

}
}

void batch_dense(int * x_in, int * x_out, int * firing_t, float * w, int N_in, int N_out, int batchSize, float th_val, int t_max){
    float * v;
    hipMalloc((void **) &v, sizeof(float)*batchSize*N_out*t_max);

    dim3 threadsPerBlock(8, 8, 16);
    dim3 blocksPerGrid(1, 1, 1);
        blocksPerGrid.z = ceil(float(batchSize)/float(threadsPerBlock.x));
        blocksPerGrid.y = ceil(float(N_out)/float(threadsPerBlock.y));
        blocksPerGrid.x = ceil(float(t_max)/float(threadsPerBlock.x));
        // cout<<blocksPerGrid.x<<'\t'<<blocksPerGrid.y<<'\t'<<blocksPerGrid.z<<endl;
    batch_tensordot<<<blocksPerGrid, threadsPerBlock>>>(x_in, w, v, batchSize, N_in, N_out, t_max);
    batch_cumsum<<<blocksPerGrid, threadsPerBlock>>>(v, N_out, t_max, batchSize);
    batch_thresholding<<<blocksPerGrid, threadsPerBlock>>>(batchSize, x_out, firing_t, v, N_out, t_max, th_val);

}
